
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/* estrutura para tratar cada pixel da imagem, ela possui as tres componentes de cada pixel (red, green e blue) */
typedef struct{
	unsigned char r;
	unsigned char g;
	unsigned char b;
}rgb;

//assinatura das funcoes
__global__ void smooth (rgb *, rgb *, int , int);
void hipError_t(hipError_t);


int main (int argc, char **argv){
	FILE *file;
	int i, rows, columns, max;
	rgb *imgH, *newImgH, *imgD, *newImgD;
    clock_t cInit, cFinal;
	
	/* abre a imagem a qual sera aplicado o filtro */
	file = fopen("in.ppm", "rb");
	fseek(file, 2, SEEK_SET);
	fscanf(file, "%d", &columns);
	fscanf(file, "%d", &rows);
	
	/* alocando memória para a matriz que irá armazenar as componentes r,g e b da imagem de entrada*/
	imgH = (rgb*) malloc ((rows*columns)*sizeof(rgb));
	
	/* alocando memória para a matriz que irá armazenar as componentes r,g e b da imagem de saída*/
	newImgH = (rgb*) malloc ((rows*columns)*sizeof(rgb));
	fscanf(file,"%d\n",&max);
	
	/* lendo a imagem do arquivo de entrada para a matriz */	
	for(i = 0; i < rows*columns; i++){
		fread(&imgH[i].r,sizeof(unsigned char),1,file);
		fread(&imgH[i].g,sizeof(unsigned char),1,file);
		fread(&imgH[i].b,sizeof(unsigned char),1,file);
	}
	fclose(file);
	
	/* aloca memoria na GPU para a imagem a ser processada */
	hipMalloc(&imgD, sizeof(rgb)*rows*columns);

	/* aloca memoria na GPU para o resultado da imagem a ser processada */
	hipError_t(hipMalloc(&newImgD, sizeof(rgb)*rows*columns));

	/* copia a matriz da imagem da memoria da CPU para a da GPU */
	hipError_t(hipMemcpy(imgD, imgH, sizeof(rgb)*rows*columns ,hipMemcpyHostToDevice));
	
	/* define o numero de threads nas dimensoes x e y por blocos e o tamanho do grid */ 
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks ((columns + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1 ) / threadsPerBlock.y);

    cInit = clock();
	/* faz a chamada da funcao que aplica o filtro em CUDA, passando os parametros definidos acima */
	smooth<<<numBlocks, threadsPerBlock>>>(imgD, newImgD, columns, rows);

	/*  garante que o host não execute até que todas as operações CUDA terminem */
	hipDeviceSynchronize();

    cFinal = clock();

	/* copia a nova imagem da memoria da GPU para a memoria da CPU */
	hipMemcpy(newImgH, newImgD, sizeof(rgb)*rows*columns ,hipMemcpyDeviceToHost);    
	
    printf("Tempo: %lf segundos\n", (double)(cFinal - cInit) / CLOCKS_PER_SEC);

	/*escreve a nova imagem */
	file = fopen("out.ppm", "wb");
	fprintf(file, "P6\n");
	fprintf(file, "%d %d\n",columns,rows);
	fprintf(file, "%d\n",max);
	for(i = 0; i < rows*columns; i++){
		fwrite(&newImgH[i].r ,sizeof(unsigned char),1,file);
		fwrite(&newImgH[i].g ,sizeof(unsigned char),1,file);
		fwrite(&newImgH[i].b ,sizeof(unsigned char),1,file);
	}
	fclose(file);

	/* liberando a memória utilizada */ 
	free(imgH);
	free(newImgH);
	hipFree(imgD);
	hipFree(newImgD);	
	return 0;	
}

/* função que retorna a média de uma componente do pixel utilizando os valores da componente ao redor dela (numa sub matriz 5x5) */
__global__ void smooth(rgb *image, rgb *newImg, int cols, int rows){
	int x, y;
	/* define o valor para x e y que ira trabalhar agora baseado na thread e no block que esta */
	x = blockIdx.y * blockDim.y + threadIdx.y; 
	y = blockIdx.x * blockDim.x + threadIdx.x;
	/* caso seja maior que o limite da imagem nao faz nada e termina */
	if(x > rows-1 || y > cols - 1)
		return;
	int i, j;
	int sumR = 0,sumG = 0,sumB = 0, count = 0;
	/* percorre os 5 pixels ao redor do ponto atual(em todas as direcoes) calculando a soma de seus valores para r,g e b */
	for(i = x-2; i < x+2; i++){
		for(j = y-2; j < y+2; j++){
			if((j < 0 || j > cols-1) || (i < 0 || i > rows-1));
			else{				
				sumR += image[i * cols + j].r;
				sumG += image[i * cols + j].g;
				sumB += image[i * cols + j].b;
				count++;
			}
		}
	}
	/* divide a soma calculada anteriomente pelo numero de pixels percorridos e escreve na matriz da nova imagem */
	newImg[x * cols + y].r = sumR/count;
	newImg[x * cols + y].g = sumG/count;
	newImg[x * cols + y].b = sumB/count;
}

/* funcao que trata dos erros que podem ocorre na execucao CUDA */
void hipError_t(hipError_t error){
	if (error != hipSuccess) {
		fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}
